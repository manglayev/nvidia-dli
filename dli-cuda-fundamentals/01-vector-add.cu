
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  size_t threads_per_block = 1024;
  size_t number_of_blocks = 64;

  addVectorsInto<<<number_of_blocks, threads_per_block>>>(c, a, b, N);
  hipDeviceSynchronize();

  hipError_t err_3;
  err_3 = hipGetLastError(); // `cudaGetLastError` will return the error from above.
  if (err_3 != hipSuccess)
  {
      printf("Error: %s\n", hipGetErrorString(err_3));
  }
  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
